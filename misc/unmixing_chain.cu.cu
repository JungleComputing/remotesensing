#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
//											Kernels SPCA													  //
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

/*Este kernel realiza la multiplicación d_a * d_X = d_y (está hecho como una reducción)
elementos_X es el lines_samples mas una serie de elementos 0s para que sea multiplo de
el numero de hilos por bloques, así podemos hacer la reduccion de forma perfecta. 

GT: This kernel performs multiplication d_a * D_X = d_y (made as a reduction)
lines_samples elementos_X is more a series of 0s elements to be a multiple of
the number of threads per block, so we can make a perfect reduction.
*/

__global__ void reduccion_yi(float *d_X,float *d_yi, float *d_a, int elementos_X)

{
	if (threadIdx.x==0){
		d_yi[blockIdx.x]=0;
	}
	int part=elementos_X/(blockDim.x*2);
	__shared__ float s_a[512];
	for(int it=0; it<part; it++){
		unsigned int tid = threadIdx.x;
		unsigned int i =(elementos_X*blockIdx.x)+ (1024*it) + threadIdx.x;
		s_a[tid]=d_X[i]*d_a[i%elementos_X]+d_X[i+blockDim.x]*d_a[(i%elementos_X)+blockDim.x];
		__syncthreads();
		for (unsigned int s = blockDim.x / 2; s > 0; s>>=1)
		{
			if (tid < s){
				s_a[tid]+=s_a[tid+s];
			}
			__syncthreads();
		}
		d_yi[blockIdx.x]+=s_a[0];
		__syncthreads();
	}
}

/*
Este kernel calcula los elementos del vector d_sum. d_sum es un vector columna, el elemento
i de s_sum se calcula como la suma de los elementos j de la fila i de la matriz d_X siempre
que el elemento j del vector d_yi sea mayor que 0.

This kernel calculates d_sum vector elements. d_sum is a column vector, the element
s_sum i is calculated as the sum of the elements j of row i of the matrix always D_X
the element j d_yi vector is greater than 0.
*/

__global__ void calcular_sum(float *d_X,float *d_yi, float *d_sum, float *d_a, int elementos_X, int num_bands){
	int idx=blockDim.x * blockIdx.x + threadIdx.x;
	__shared__ float s_sum[32];
	s_sum[threadIdx.x]=0;
	for (int i=0; i<num_bands; i++){
		if(d_yi[i]>0){
			s_sum[threadIdx.x]+=d_X[i*elementos_X+idx];
		}
	}
	__syncthreads();
	d_sum[blockIdx.x*blockDim.x+threadIdx.x]=s_sum[threadIdx.x];
}


/*

Este kernel realiza la multiplicación d_a * d_X = d_aux (está hecho como una reducción).
elementos_X es el lines_samples mas una serie de elementos 0s para que sea multiplo de el
numero de hilos por bloques, así podemos hacer la reduccion de forma perfecta. 

This kernel performs multiplication d_a * D_X = d_aux (made as a reduction).
lines_samples elementos_X is more a series of 0s elements to be a multiple of the
number of threads per block, so we can make perfectly reduction
*/

__global__ void reduccion_aux(float *d_X,float *d_aux, float *d_a, int elementos_X){
	int part=elementos_X/(blockDim.x*2);
	__shared__ float s_a[512];
	for(int it=0; it<part; it++){
		unsigned int tid = threadIdx.x;
		unsigned int i =(elementos_X*blockIdx.x)+ (1024*it) + threadIdx.x;
		s_a[tid]=d_X[i]*d_a[i%elementos_X]+d_X[i+blockDim.x]*d_a[(i%elementos_X)+blockDim.x];
		__syncthreads();
		for (unsigned int s = blockDim.x / 2; s > 0; s>>=1)
		{
			if (tid < s){
				s_a[tid]+=s_a[tid+s];
			}
			__syncthreads();
		}
		d_aux[blockIdx.x]+=s_a[0];
		__syncthreads();
	}
}

/*

Este kernel modifica la matriz X restándole a cada elemento el producto de los vectores
d_aux y d_a (el resultado es un número).

This kernel modifies the array to each element X minus the product of vectors
d_aux and d_a (the result is a number).
*/

__global__ void calcular_X(float *d_X,float *d_aux, float *d_a, int elementos_X, int num_bands){
	int idx=blockDim.x * blockIdx.x + threadIdx.x;
	__shared__ float s_a[32];
	s_a[threadIdx.x]=d_a[idx];
	for (int i=0; i<num_bands; i++){
			d_X[i*elementos_X+idx]-=s_a[threadIdx.x]*d_aux[i];
	}
}

////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
// Kernels NFINDR	                                                              //
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

#define N 19
#define ABS(a)	   (((a) < 0) ? -(a) : (a))

/*
Este kernel calcula el volumen conseguido al agregar cada pixel al conjunto de endmember
Realiza la multiplicación de d_aux * d_HIM2x2, ademas calcula el valor absoluto de cada volumen.

This kernel calculates the volume achieved by adding each pixel to the set of endmembers.
Performs multiplication of d_aux * d_HIM2x2, also calculates the absolute value of each volume.
*/
__global__ void CalcularVolumenes(double *d_aux, double *d_HIM2x2, double *d_Vvolume,double tmp2,int lines_samples)

{
	int idx =  blockDim.x * blockIdx.x+threadIdx.x;
	__shared__ double s_aux[N];
	double a;
	if (idx<lines_samples){
		if(threadIdx.x<N){
			s_aux[threadIdx.x]=d_aux[threadIdx.x];
		}
		syncthreads();
		a=0;
		for(int i=0; i<N; i++){
			a+=s_aux[i]*d_HIM2x2[i*lines_samples+idx];
		}
		a=a*tmp2;
		ABS(a);
		d_Vvolume[idx]=a;	
	}
}

/*
Este kernel obtiene los I volumenes mayores calculados en el kernel anterior siendo I el número
de bloques con que se estructura el lanzamiento del kernel. Además obtiene los índices de los pixel
que otienen dichos volumenes.

This kernel I get the older volumes calculated in the previous kernel I being the number
blocks with which the release of kernel structure. He also gets pixel indices
Otieno said that volumes.
*/

__global__ void ReduccionVolumenes(double *d_Vvolume, double *d_volumenes, int *d_indices){

	__shared__ double s_v[512];
	__shared__ int s_i[512];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	if((i+blockDim.x)>=122500){
			s_v[tid]=d_Vvolume[i];
			s_i[tid]=i;
	}
	else{
		if(d_Vvolume[i]>d_Vvolume[i + blockDim.x]){
			s_v[tid]=d_Vvolume[i];
			s_i[tid]=i;
		}
		else{
			s_v[tid]=d_Vvolume[i + blockDim.x];
			s_i[tid]=i+ blockDim.x;
		}
	}
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s>>=1){
		if (tid < s){
			if(s_v[tid]<=s_v[tid+s]){
				s_v[tid]=s_v[tid+s];
				s_i[tid]=s_i[tid+s];
			}
		}
		__syncthreads();
	}
	d_volumenes[blockIdx.x]=s_v[0];
	d_indices[blockIdx.x]=s_i[0];
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
// Kernels Unmixing                                                                   //
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

#define TAMANIO_MATRIZ_C 188

/*
Este kernel realiza la fase final del unmixing, es decir multiplicar cada pixel por la
matriz de cómputo obtenida a partir de la matriz de endmembers, y así obtener las abundancias.

This kernel performs the final stage of unmixing, ie multiply each pixel by
computing matrix obtained from the endmembers matrix, and obtain the abundances.
*/
__global__ void Unmixing(float *d_imagen, float *d_imagen_unmixing,float *d_matriz_computo, int num_lines, int num_samples, int num_bands, int N_END)

{
	int pixel =  blockDim.x * blockIdx.x+threadIdx.x;
	
	__shared__ float matriz_c[TAMANIO_MATRIZ_C];
	float l_pixel[188];
	float a;
	if(pixel<num_lines*num_samples){
		for(int t=0; t<num_bands; t++){
			l_pixel[t]=d_imagen[pixel+(num_lines*num_samples*t)];
		}
		for(int it=0; it<N_END; it++){
			if(threadIdx.x==0){
				for(int i=0; i<num_bands; i++){
					matriz_c[i]=d_matriz_computo[it*num_bands+i];
				}
			}
			syncthreads();
			a=0;
			for(int k=0; k<num_bands; k++){	
				a+=matriz_c[k]*l_pixel[k];
			}
			d_imagen_unmixing[pixel+(num_lines*num_samples*it)]=a;
		}
	}

}
